#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <random>
#include <algorithm>
#include <vector>
#include <unordered_map>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thread>

__global__ void findUniqueNumbers
(
    const int* arrayPtr,
    int* uniqueNumbersPtr,
    int* uniqueCountersPtr
)
{
    const int ARRAY_SIZE = 10000000;
    const int BLOCK_ARRAY_SIZE = 10000;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    extern __shared__ int sharedSet[];

    for (int i = tid; i < ARRAY_SIZE; i += stride)
    {
        int value = arrayPtr[i];

        bool isPrevSame = (0 < i && value == arrayPtr[i - 1]);
        bool isNextSame = (i < ARRAY_SIZE - 1 && value == arrayPtr[i + 1]);

        bool isUnique = !(isPrevSame || isNextSame);

        int uniqueIndex = atomicAdd(&uniqueCountersPtr[blockIdx.x], isUnique);

        sharedSet[isUnique * uniqueIndex + !isUnique * BLOCK_ARRAY_SIZE] = value;
    }

    __syncthreads();

    if (threadIdx.x != 0) return;

    for (int i = 0; i < uniqueCountersPtr[blockIdx.x]; ++i)
    {
        uniqueNumbersPtr[blockIdx.x * BLOCK_ARRAY_SIZE + i] = sharedSet[i];
    }
}

void createArray
(
    std::vector<int>* finalValues, 
    int* uniqueValuesAmount, 
    const int arraySize, 
    const int uniqueMaxAmount, 
    const int uniqueValuesRange
)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> uniqueValuesAmountDis(1, uniqueMaxAmount);
    std::uniform_int_distribution<int> valuesDis(0, uniqueValuesRange);

    *uniqueValuesAmount = uniqueValuesAmountDis(gen);

    std::vector<int> uniqueSet;
    for (int i = 0; i < *uniqueValuesAmount; ++i) {
        int value;
        do
        {
            value = valuesDis(gen);
        } while (std::find(uniqueSet.begin(), uniqueSet.end(), value) != uniqueSet.end());
        uniqueSet.push_back(value);
    }
    std::cout << "Unique Amount: " << uniqueSet.size() << " ";
    std::cout << "GeneratedUnique: ";
    std::sort(uniqueSet.begin(), uniqueSet.end());
    for (int i : uniqueSet) {
        std::cout << i << " ";
    }
    std::cout << std::endl;

    int randomValuesSize = arraySize - *uniqueValuesAmount;
    std::vector<int> randomSet;
    for (int i = 0; i < arraySize - *uniqueValuesAmount; ++i) {
        int value;
        do
        {
            value = valuesDis(gen);
        } while (std::find(uniqueSet.begin(), uniqueSet.end(), value) != uniqueSet.end());
        randomSet.push_back(value);
    }

    finalValues->reserve(arraySize);
    finalValues->insert(finalValues->end(), uniqueSet.begin(), uniqueSet.end());
    finalValues->insert(finalValues->end(), randomSet.begin(), randomSet.end());
}

int main() {
    const int arraySize = 10000000;
    const int uniqueMaxAmount = 1000;
    const int uniqueValuesRange = 10000;

    // CPU

    int uniqueValuesAmount = 0;
    std::vector<int> finalValues;
    createArray(&finalValues, &uniqueValuesAmount, arraySize, uniqueMaxAmount, uniqueValuesRange);

    size_t streamsAmount = finalValues.size() / 10000; // 1000

    // GPU

    thrust::sort(thrust::host, finalValues.begin(), finalValues.end());

    int* gpuArrayPtr;
    int* gpuUniqueNumbersPtr;
    int* gpuUniqueCountersPtr;

    hipMalloc((void**)&gpuArrayPtr, arraySize * sizeof(int));
    hipMalloc((void**)&gpuUniqueNumbersPtr, arraySize * sizeof(int));
    hipMalloc((void**)&gpuUniqueCountersPtr, streamsAmount * sizeof(int));

    hipMemcpy(gpuArrayPtr, &finalValues.at(0), arraySize * sizeof(int), hipMemcpyHostToDevice);

    int numBlock = streamsAmount;
    int numThreads = 512;

    findUniqueNumbers
        << <
        numBlock,
        numThreads,
        (arraySize / streamsAmount + 1) * sizeof(int) // Shared memory per block
        >> >
        (
            gpuArrayPtr,
            gpuUniqueNumbersPtr,
            gpuUniqueCountersPtr
            );

    hipDeviceSynchronize();

    std::vector<int> finalVl;

    for (int blockIndex = 0; blockIndex < streamsAmount; ++blockIndex)
    {
        int hostUniqueCount = 0;
        hipMemcpy(&hostUniqueCount, (gpuUniqueCountersPtr + blockIndex), sizeof(int), hipMemcpyDeviceToHost);

        std::vector<int> hostUniqueNumbers(hostUniqueCount);

        hipMemcpy(hostUniqueNumbers.data(), (gpuUniqueNumbersPtr + 10000 * blockIndex), hostUniqueCount * sizeof(int), hipMemcpyDeviceToHost);

        finalVl.insert(finalVl.end(), hostUniqueNumbers.begin(), hostUniqueNumbers.end());
    }

    hipFree(gpuArrayPtr);
    hipFree(gpuUniqueNumbersPtr);
    hipFree(gpuUniqueCountersPtr);

    std::cout << "Unique Amount: " << finalVl.size() << " ";
    std::cout << "   Found Unique: ";
    std::sort(finalVl.begin(), finalVl.end());

    for (auto o : finalVl)
    {
        std::cout << o << " ";
    }

    return 0;
}